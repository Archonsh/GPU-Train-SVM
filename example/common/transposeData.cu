
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BSIZE 256

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

__global__ void transpose(int nRows, int nCols, float* devData, float* devTranData) {
	int pointId = threadIdx.x;

	while(pointId < nRows) {
		int inPos = pointId*nCols;
		int i;
		for(i = 0; i < nCols; i++)
			devTranData[pointId+i*nRows] = devData[inPos+i];

		pointId += BSIZE;
	}

}

void transposeData(int nRows, int nCols, float* data, float* transposedData) {
	float* devData;
	CHECK(hipMalloc((float**)&devData, sizeof(float)*nRows*nCols));
	CHECK(hipMemcpy(devData, data, sizeof(float)*nRows*nCols, hipMemcpyHostToDevice));

	float* devTranData;
	CHECK(hipMalloc((float**)&devTranData, sizeof(float)*nRows*nCols));

	transpose<<<1, BSIZE>>>(nRows, nCols, devData, devTranData);

	CHECK(hipMemcpy(transposedData, devTranData, sizeof(float)*nRows*nCols, hipMemcpyDeviceToHost));

	CHECK(hipFree(devData));
	CHECK(hipFree(devTranData));
}